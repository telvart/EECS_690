#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello(){
  printf("Hello, CUDA!\n");
}

void run()
{
  hello<<<1,10>>>();
  hipDeviceSynchronize();
}
