#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "proj3CUDA.h"

__global__ void hello(){
  printf("Hello, CUDA!\n");
}

void run()
{
  hello<<<1,1>>>();
  hipDeviceSynchronize();
}
